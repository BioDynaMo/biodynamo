#include "hip/hip_runtime.h"
#include "samples/common/inc/hip/hip_vector_types.h"
#include "gpu/displacement_op_cuda_kernel.h"

#define GpuErrchk(ans) { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (code == hipErrorInsufficientDriver) {
        printf("This probably means that no CUDA-compatible GPU has been detected. Consider setting the use_opencl flag to \"true\" in the bmd.toml file to use OpenCL instead.\n");
      }
      if (abort) exit(code);
   }
}

// __device__ float squared_euclidian_distance(float3 my_position, float3 nb_position) {
//   const float dx = positions[3*idx + 0] - positions[3*nidx + 0];
//   const float dy = positions[3*idx + 1] - positions[3*nidx + 1];
//   const float dz = positions[3*idx + 2] - positions[3*nidx + 2];
//   return (dx * dx + dy * dy + dz * dz);
// }

__device__ int3 get_box_coordinates(float3 pos, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords;
  box_coords.x = (floor(pos.x) - grid_dimensions[0]) / box_length;
  box_coords.y = (floor(pos.y) - grid_dimensions[1]) / box_length;
  box_coords.z = (floor(pos.z) - grid_dimensions[2]) / box_length;
  return box_coords;
}

__device__ int3 get_box_coordinates_2(uint32_t box_idx, uint32_t* num_boxes_axis_) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis_[0]*num_boxes_axis_[1]);
  uint32_t remainder = box_idx % (num_boxes_axis_[0]*num_boxes_axis_[1]);
  box_coord.y = remainder / num_boxes_axis_[0];
  box_coord.x = remainder % num_boxes_axis_[0];
  return box_coord;
}

__device__ uint32_t get_box_id_2(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0]*num_boxes_axis[1] + bc.y * num_boxes_axis[0] + bc.x;
}

__device__ uint32_t get_box_id(float3 pos, uint32_t* num_boxes_axis, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length);
  return get_box_id_2(box_coords, num_boxes_axis);
}

__device__ void compute_force(const float3& my_position, float my_diameter, float* positions, float* diameters, uint32_t nidx, float3& result) {
  float r1 = 0.5 * my_diameter;
  float r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a desired density.
  float additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  float3 comp;
  comp.x = my_position.x - positions[3*nidx + 0];
  comp.y = my_position.y - positions[3*nidx + 1];
  comp.z = my_position.z - positions[3*nidx + 2];
  float center_distance = length(comp);

  // the overlap distance (how much one penetrates in the other)
  float delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    result += make_float3(42.0, 42.0, 42.0);
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  float r = (r1 * r2) / (r1 + r2);
  float gamma = 1; // attraction coeff
  float k = 2;     // repulsion coeff
  float f = k * delta - gamma * sqrt(r * delta);

  float module = f / center_distance;
  result += module*comp;
  // result.x += module * comp.x;
  // result.y += module * comp.y;
  // result.z += module * comp.z;
  // printf("%f, %f, %f\n", module * comp1, module * comp2, module * comp3);
  // printf("Force between cell (%u) [%f, %f, %f] & cell (%u) [%f, %f, %f] = %f, %f, %f\n", idx, positions[3*idx + 0], positions[3*idx + 1], positions[3*idx + 2], nidx, positions[3*nidx + 0], positions[3*nidx + 1], positions[3*nidx + 2], module * comp1, module * comp2, module * comp3);
}

__device__ void GetMooreBoxIds(uint32_t box_idx, uint32_t* ret, uint32_t* num_boxes_axis) {
  const int3 moore_offset[27] = {
    make_int3(-1, -1, -1), make_int3(0, -1, -1), make_int3(1, -1, -1),
    make_int3(-1, 0, -1),  make_int3(0, 0, -1),  make_int3(1, 0, -1),
    make_int3(-1, 1, -1),  make_int3(0, 1, -1),  make_int3(1, 1, -1),
    make_int3(-1, -1, 0),  make_int3(0, -1, 0),  make_int3(1, -1, 0),
    make_int3(-1, 0, 0),   make_int3(0, 0, 0),   make_int3(1, 0, 0),
    make_int3(-1, 1, 0),   make_int3(0, 1, 0),   make_int3(1, 1, 0),
    make_int3(-1, -1, 1),  make_int3(0, -1, 1),  make_int3(1, -1, 1),
    make_int3(-1, 0, 1),   make_int3(0, 0, 1),   make_int3(1, 0, 1),
    make_int3(-1, 1, 1),   make_int3(0, 1, 1),   make_int3(1, 1, 1)};

  int3 box_coords = get_box_coordinates_2(box_idx, num_boxes_axis);
  for (unsigned i = 0; i < 27; i++) {
    ret[i] = get_box_id_2(box_coords + moore_offset[i], num_boxes_axis);
  }
}

__constant__ bdm::SimParams params;

__global__ void collide(
       float* positions,
       float* diameters,
       float* tractor_force,
       float* adherence,
       uint32_t* box_id,
       float* mass,
       uint32_t* starts,
       uint16_t* lengths,
       uint32_t* successors,
       float* result) {
  __shared__ uint32_t moore_boxes[27];
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < params.num_objects) {
    float3 collision_force = make_float3(
                                params.timestep * tractor_force[3*tidx + 0],
                                params.timestep * tractor_force[3*tidx + 1],
                                params.timestep * tractor_force[3*tidx + 2]);

    float3 my_position;
    my_position = make_float3(positions[3 * tidx], positions[3 * tidx + 1],
      positions[3 * tidx + 2]);
    float my_diameter = diameters[tidx];

    GetMooreBoxIds(box_id[tidx], &moore_boxes[0], params.num_boxes_axis);
    for (int i = 0; i < 27; i++) {
      uint32_t bidx = moore_boxes[i];
      uint32_t nidx = starts[bidx];
      for (uint16_t nb = 0; nb < lengths[bidx]; nb++) {
        if (nidx != tidx) {
          if (dot(my_position, make_float3(positions[3 * nidx], positions[3 * nidx + 1],
            positions[3 * nidx + 2])) < params.squared_radius) {
            compute_force(my_position, my_diameter, positions, diameters, nidx, collision_force);
          }
        }
        // traverse linked-list
        nidx = successors[nidx];
      }
    }

    // Mass needs to non-zero!
    float mh = params.timestep / mass[tidx];

    if (length(collision_force) > adherence[tidx]) {
      result[3*tidx + 0] = collision_force.x * mh;
      result[3*tidx + 1] = collision_force.y * mh;
      result[3*tidx + 2] = collision_force.z * mh;

      if (length(collision_force) * mh > params.max_displacement) {
        result[3*tidx + 0] = params.max_displacement;
        result[3*tidx + 1] = params.max_displacement;
        result[3*tidx + 2] = params.max_displacement;
      }
    }
  }
}

bdm::DisplacementOpCudaKernel::DisplacementOpCudaKernel(uint32_t num_objects, uint32_t num_boxes) {
  GpuErrchk(hipMalloc(&d_positions_, 3 * num_objects * sizeof(float)));
  GpuErrchk(hipMalloc(&d_diameters_, num_objects * sizeof(float)));
  GpuErrchk(hipMalloc(&d_tractor_force_, 3 * num_objects * sizeof(float)));
  GpuErrchk(hipMalloc(&d_adherence_, num_objects * sizeof(float)));
  GpuErrchk(hipMalloc(&d_box_id_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_mass_, num_objects * sizeof(float)));
  GpuErrchk(hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t)));
  GpuErrchk(hipMalloc(&d_successors_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_cell_movements_, 3 * num_objects * sizeof(float)));
}

void bdm::DisplacementOpCudaKernel::LaunchDisplacementKernel(float* positions, float* diameters, float* tractor_force,
                    float* adherence, uint32_t* box_id, float* mass,
                    uint32_t* starts, uint16_t* lengths, uint32_t* successors,
                    float* cell_movements, SimParams host_params) {
  uint32_t num_boxes = host_params.num_boxes_axis[0] * host_params.num_boxes_axis[1] * host_params.num_boxes_axis[2];

  GpuErrchk(hipMemcpy(d_positions_, 		positions, 3 * host_params.num_objects * sizeof(float), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_diameters_, 		diameters, host_params.num_objects * sizeof(float), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_tractor_force_, 	tractor_force, 3 * host_params.num_objects * sizeof(float), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_adherence_,     adherence, host_params.num_objects * sizeof(float), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_box_id_, 		box_id, host_params.num_objects * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_mass_, 				mass, host_params.num_objects * sizeof(float), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_starts_, 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_lengths_, 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_successors_, 		successors, host_params.num_objects * sizeof(uint32_t), hipMemcpyHostToDevice));

  hipMemcpyToSymbol(HIP_SYMBOL(params), &host_params, sizeof(SimParams));

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0, host_params.num_objects);
  gridSize = (host_params.num_objects + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  collide<<<gridSize, blockSize>>>(d_positions_, d_diameters_, d_tractor_force_,
    d_adherence_, d_box_id_, d_mass_, d_starts_, d_lengths_, d_successors_,
    d_cell_movements_);

  // We need to wait for the kernel to finish before reading back the result
  hipDeviceSynchronize();
  hipMemcpy(cell_movements, d_cell_movements_, 3 * host_params.num_objects * sizeof(float), hipMemcpyDeviceToHost);
}

void bdm::DisplacementOpCudaKernel::ResizeCellBuffers(uint32_t num_cells) {
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_successors_);
  hipFree(d_cell_movements_);

  hipMalloc(&d_positions_, 3 * num_cells * sizeof(float));
  hipMalloc(&d_diameters_, num_cells * sizeof(float));
  hipMalloc(&d_tractor_force_, 3 * num_cells * sizeof(float));
  hipMalloc(&d_adherence_, num_cells * sizeof(float));
  hipMalloc(&d_box_id_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_mass_, num_cells * sizeof(float));
  hipMalloc(&d_successors_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_cell_movements_, 3 * num_cells * sizeof(float));
}

void bdm::DisplacementOpCudaKernel::ResizeGridBuffers(uint32_t num_boxes) {
  hipFree(d_starts_);
  hipFree(d_lengths_);

  hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t));
  hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t));
}

bdm::DisplacementOpCudaKernel::~DisplacementOpCudaKernel() {
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_starts_);
  hipFree(d_lengths_);
  hipFree(d_successors_);
  hipFree(d_cell_movements_);
}
