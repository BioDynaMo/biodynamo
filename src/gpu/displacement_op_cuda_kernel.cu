#include "hip/hip_runtime.h"
#include "hip/hip_vector_types.h"
#include "gpu/displacement_op_cuda_kernel.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (code == hipErrorInsufficientDriver) {
        printf("This probably means that no CUDA-compatible GPU has been detected. Consider setting the use_opencl flag to \"true\" in the bmd.toml file to use OpenCL instead.\n");
      }
      if (abort) exit(code);
   }
}

__device__ double norm(double3 v) {
  return sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
}

__device__ double squared_euclidian_distance(double* positions, uint32_t idx, uint32_t nidx) {
  const double dx = positions[3*idx + 0] - positions[3*nidx + 0];
  const double dy = positions[3*idx + 1] - positions[3*nidx + 1];
  const double dz = positions[3*idx + 2] - positions[3*nidx + 2];
  return (dx * dx + dy * dy + dz * dz);
}

__device__ int3 get_box_coordinates(double3 pos, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords;
  box_coords.x = (floor(pos.x) - grid_dimensions[0]) / box_length;
  box_coords.y = (floor(pos.y) - grid_dimensions[1]) / box_length;
  box_coords.z = (floor(pos.z) - grid_dimensions[2]) / box_length;
  return box_coords;
}

__device__ int3 get_box_coordinates_2(uint32_t box_idx, uint32_t* num_boxes_axis_) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis_[0]*num_boxes_axis_[1]);
  uint32_t remainder = box_idx % (num_boxes_axis_[0]*num_boxes_axis_[1]);
  box_coord.y = remainder / num_boxes_axis_[0];
  box_coord.x = remainder % num_boxes_axis_[0];
  return box_coord;
}

__device__ uint32_t get_box_id_2(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0]*num_boxes_axis[1] + bc.y * num_boxes_axis[0] + bc.x;
}

__device__ uint32_t get_box_id(double3 pos, uint32_t* num_boxes_axis, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length);
  return get_box_id_2(box_coords, num_boxes_axis);
}

__device__ void compute_force(double* positions, double* diameters, uint32_t idx, uint32_t nidx, double3* result) {
  double r1 = 0.5 * diameters[idx];
  double r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a desired density.
  double additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  double comp1 = positions[3*idx + 0] - positions[3*nidx + 0];
  double comp2 = positions[3*idx + 1] - positions[3*nidx + 1];
  double comp3 = positions[3*idx + 2] - positions[3*nidx + 2];
  double center_distance = sqrt(comp1 * comp1 + comp2 * comp2 + comp3 * comp3);

  // the overlap distance (how much one penetrates in the other)
  double delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    result->x += 42.0;
    result->y += 42.0;
    result->z += 42.0;
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  double r = (r1 * r2) / (r1 + r2);
  double gamma = 1; // attraction coeff
  double k = 2;     // repulsion coeff
  double f = k * delta - gamma * sqrt(r * delta);

  double module = f / center_distance;
  result->x += module * comp1;
  result->y += module * comp2;
  result->z += module * comp3;
  // printf("%f, %f, %f\n", module * comp1, module * comp2, module * comp3);
  // printf("Force between cell (%u) [%f, %f, %f] & cell (%u) [%f, %f, %f] = %f, %f, %f\n", idx, positions[3*idx + 0], positions[3*idx + 1], positions[3*idx + 2], nidx, positions[3*nidx + 0], positions[3*nidx + 1], positions[3*nidx + 2], module * comp1, module * comp2, module * comp3);
}

__device__ void default_force(double* positions,
                   double* diameters,
                   uint32_t idx, uint32_t start, uint16_t length,
                   uint32_t* successors,
                   double* squared_radius,
                   double3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (squared_euclidian_distance(positions, idx, nidx) < squared_radius[0]) {
        compute_force(positions, diameters, idx, nidx, result);
      }
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__global__ void collide(
       double* positions,
       double* diameters,
       double* tractor_force,
       double* adherence,
       uint32_t* box_id,
       double* mass,
       double* timestep,
       double* max_displacement,
       double* squared_radius,
       uint32_t* N,
       uint32_t* starts,
       uint16_t* lengths,
       uint32_t* successors,
       uint32_t* box_length,
       uint32_t* num_boxes_axis,
       int32_t* grid_dimensions,
       double* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < N[0]) {
    result[3*tidx + 0] = timestep[0] * tractor_force[3*tidx + 0];
    result[3*tidx + 1] = timestep[0] * tractor_force[3*tidx + 1];
    result[3*tidx + 2] = timestep[0] * tractor_force[3*tidx + 2];
    // printf("cell_movement = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);
    
    double3 collision_force = make_double3(0, 0, 0);

    // Moore neighborhood
    int3 box_coords = get_box_coordinates_2(box_id[tidx], num_boxes_axis);
    for (int z = -1; z <= 1; z++) {
      for (int y = -1; y <= 1; y++) {
        for (int x = -1; x <= 1; x++) {
          uint32_t bidx = get_box_id_2(box_coords + make_int3(x, y, z), num_boxes_axis);
          if (lengths[bidx] != 0) {
            default_force(positions, diameters, tidx, starts[bidx], lengths[bidx], successors, squared_radius, &collision_force);
          }
        }
      }
    }

    // Mass needs to non-zero!
    double mh = timestep[0] / mass[tidx];
    // printf("mh = %f\n", mh);

    if (norm(collision_force) > adherence[tidx]) {
      result[3*tidx + 0] += collision_force.x * mh;
      result[3*tidx + 1] += collision_force.y * mh;
      result[3*tidx + 2] += collision_force.z * mh;
      // printf("collision_force = (%f, %f, %f)\n", collision_force.x, collision_force.y, collision_force.z);
      // printf("cell_movement (1) = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);

      if (norm(collision_force) * mh > max_displacement[0]) {
        result[3*tidx + 0] = max_displacement[0];
        result[3*tidx + 1] = max_displacement[0];
        result[3*tidx + 2] = max_displacement[0];
      }
      // printf("cell_movement (2) = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);
    }
  }
}

bdm::DisplacementOpCudaKernel::DisplacementOpCudaKernel(uint32_t N, uint32_t num_boxes) {
  // printf("N = %u  |  num_boxes = %u\n", N, num_boxes);
  gpuErrchk(hipMalloc(&d_positions, 3 * N * sizeof(double)));
  gpuErrchk(hipMalloc(&d_diameters, N * sizeof(double)));
  gpuErrchk(hipMalloc(&d_tractor_force, 3 * N * sizeof(double)));
  gpuErrchk(hipMalloc(&d_adherence, N * sizeof(double)));
  gpuErrchk(hipMalloc(&d_box_id, N * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_mass, N * sizeof(double)));
  gpuErrchk(hipMalloc(&d_timestep, sizeof(double)));
  gpuErrchk(hipMalloc(&d_max_displacement, sizeof(double)));
  gpuErrchk(hipMalloc(&d_squared_radius, sizeof(double)));
  gpuErrchk(hipMalloc(&d_N, sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_starts, num_boxes * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_lengths, num_boxes * sizeof(uint16_t)));
  gpuErrchk(hipMalloc(&d_successors, N * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_box_length, sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_num_boxes_axis, 3 * sizeof(uint32_t)));
  gpuErrchk(hipMalloc(&d_grid_dimensions, 3 * sizeof(int32_t)));
  gpuErrchk(hipMalloc(&d_cell_movements, 3 * N * sizeof(double)));
}

void bdm::DisplacementOpCudaKernel::displacement_op_cuda(double* positions, double* diameters, double* tractor_force, double* adherence, uint32_t* box_id, double* mass, double* timestep, double* max_displacement, double* squared_radius, uint32_t* N, uint32_t* starts, uint16_t* lengths, uint32_t* successors, uint32_t* box_length, uint32_t* num_boxes_axis, int32_t* grid_dimensions, double* cell_movements) {
  uint32_t num_boxes = num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

  gpuErrchk(hipMemcpy(d_positions, 		positions, 3 * N[0] * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_diameters, 		diameters, N[0] * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_tractor_force, 	tractor_force, 3 * N[0] * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_adherence,     adherence, N[0] * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_box_id, 		box_id, N[0] * sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_mass, 				mass, N[0] * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_timestep, 			timestep, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_max_displacement,  max_displacement, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_squared_radius, 	squared_radius, sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_N, 				N, sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_starts, 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_lengths, 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_successors, 		successors, N[0] * sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_box_length, 		box_length, sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_num_boxes_axis, 	num_boxes_axis, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_grid_dimensions, 	grid_dimensions, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0, N[0]);
  gridSize = (N[0] + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  collide<<<gridSize, blockSize>>>(d_positions, d_diameters, d_tractor_force, d_adherence, d_box_id, d_mass, d_timestep, d_max_displacement, d_squared_radius, d_N, d_starts, d_lengths, d_successors, d_box_length, d_num_boxes_axis, d_grid_dimensions, d_cell_movements);

  // We need to wait for the kernel to finish before reading back the result
  hipDeviceSynchronize();
  hipMemcpy(cell_movements, d_cell_movements, 3 * N[0] * sizeof(double), hipMemcpyDeviceToHost);
}

void bdm::DisplacementOpCudaKernel::resize_cell_buffers(uint32_t N) {
  hipFree(d_positions);
  hipFree(d_diameters);
  hipFree(d_tractor_force);
  hipFree(d_adherence);
  hipFree(d_box_id);
  hipFree(d_mass);
  hipFree(d_successors);
  hipFree(d_cell_movements);

  hipMalloc(&d_positions, 3 * N * sizeof(double));
  hipMalloc(&d_diameters, N * sizeof(double));
  hipMalloc(&d_tractor_force, 3 * N * sizeof(double));
  hipMalloc(&d_adherence, N * sizeof(double));
  hipMalloc(&d_box_id, N * sizeof(uint32_t));
  hipMalloc(&d_mass, N * sizeof(double));
  hipMalloc(&d_successors, N * sizeof(uint32_t));
  hipMalloc(&d_cell_movements, 3 * N * sizeof(double));
}

void bdm::DisplacementOpCudaKernel::resize_grid_buffers(uint32_t num_boxes) {
  hipFree(d_starts);
  hipFree(d_lengths);

  hipMalloc(&d_starts, num_boxes * sizeof(uint32_t));
  hipMalloc(&d_lengths, num_boxes * sizeof(uint16_t));
}

bdm::DisplacementOpCudaKernel::~DisplacementOpCudaKernel() {
  hipFree(d_positions);
  hipFree(d_diameters);
  hipFree(d_tractor_force);
  hipFree(d_adherence);
  hipFree(d_box_id);
  hipFree(d_mass);
  hipFree(d_timestep);
  hipFree(d_max_displacement);
  hipFree(d_squared_radius);
  hipFree(d_N);
  hipFree(d_starts);
  hipFree(d_lengths);
  hipFree(d_successors);
  hipFree(d_num_boxes_axis);
  hipFree(d_grid_dimensions);
  hipFree(d_cell_movements);
}
