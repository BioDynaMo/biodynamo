#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//
// Copyright (C) The BioDynaMo Project.
// All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include "samples/common/inc/hip/hip_vector_types.h"
#include "core/gpu/displacement_op_cuda_kernel.h"

#define GpuErrchk(ans) { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (code == hipErrorInsufficientDriver) {
        printf("This probably means that no CUDA-compatible GPU has been detected. Consider setting the use_opencl flag to \"true\" in the bmd.toml file to use OpenCL instead.\n");
      }
      if (abort) exit(code);
   }
}

__device__ double norm(double3 v) {
  return sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
}

__device__ double squared_euclidian_distance(double* positions, uint32_t idx, uint32_t nidx) {
  const double dx = positions[3*idx + 0] - positions[3*nidx + 0];
  const double dy = positions[3*idx + 1] - positions[3*nidx + 1];
  const double dz = positions[3*idx + 2] - positions[3*nidx + 2];
  return (dx * dx + dy * dy + dz * dz);
}

__device__ int3 get_box_coordinates(double3 pos, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords;
  box_coords.x = (floor(pos.x) - grid_dimensions[0]) / box_length;
  box_coords.y = (floor(pos.y) - grid_dimensions[1]) / box_length;
  box_coords.z = (floor(pos.z) - grid_dimensions[2]) / box_length;
  return box_coords;
}

__device__ int3 get_box_coordinates_2(uint32_t box_idx, uint32_t* num_boxes_axis_) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis_[0]*num_boxes_axis_[1]);
  uint32_t remainder = box_idx % (num_boxes_axis_[0]*num_boxes_axis_[1]);
  box_coord.y = remainder / num_boxes_axis_[0];
  box_coord.x = remainder % num_boxes_axis_[0];
  return box_coord;
}

__device__ uint32_t get_box_id_2(int3 bc, uint32_t* num_boxes_axis) {
  return bc.z * num_boxes_axis[0]*num_boxes_axis[1] + bc.y * num_boxes_axis[0] + bc.x;
}

__device__ uint32_t get_box_id(double3 pos, uint32_t* num_boxes_axis, int32_t* grid_dimensions, uint32_t box_length) {
  int3 box_coords = get_box_coordinates(pos, grid_dimensions, box_length);
  return get_box_id_2(box_coords, num_boxes_axis);
}

__device__ void compute_force(double* positions, double* diameters, uint32_t idx, uint32_t nidx, double3* result) {
  double r1 = 0.5 * diameters[idx];
  double r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a desired density.
  double additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  double comp1 = positions[3*idx + 0] - positions[3*nidx + 0];
  double comp2 = positions[3*idx + 1] - positions[3*nidx + 1];
  double comp3 = positions[3*idx + 2] - positions[3*nidx + 2];
  double center_distance = sqrt(comp1 * comp1 + comp2 * comp2 + comp3 * comp3);

  // the overlap distance (how much one penetrates in the other)
  double delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    result->x += 42.0;
    result->y += 42.0;
    result->z += 42.0;
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  double r = (r1 * r2) / (r1 + r2);
  double gamma = 1; // attraction coeff
  double k = 2;     // repulsion coeff
  double f = k * delta - gamma * sqrt(r * delta);

  double module = f / center_distance;
  result->x += module * comp1;
  result->y += module * comp2;
  result->z += module * comp3;
  // printf("%f, %f, %f\n", module * comp1, module * comp2, module * comp3);
  // printf("Force between cell (%u) [%f, %f, %f] & cell (%u) [%f, %f, %f] = %f, %f, %f\n", idx, positions[3*idx + 0], positions[3*idx + 1], positions[3*idx + 2], nidx, positions[3*nidx + 0], positions[3*nidx + 1], positions[3*nidx + 2], module * comp1, module * comp2, module * comp3);
}

__device__ void default_force(double* positions,
                   double* diameters,
                   uint32_t idx, uint32_t start, uint16_t length,
                   uint32_t* successors,
                   double* squared_radius,
                   double3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (squared_euclidian_distance(positions, idx, nidx) < squared_radius[0]) {
        compute_force(positions, diameters, idx, nidx, result);
      }
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__global__ void collide(
       double* positions,
       double* diameters,
       double* tractor_force,
       double* adherence,
       uint32_t* box_id,
       double* mass,
       double* timestep,
       double* max_displacement,
       double* squared_radius,
       uint32_t* num_objects,
       uint32_t* starts,
       uint16_t* lengths,
       uint32_t* successors,
       uint32_t* box_length,
       uint32_t* num_boxes_axis,
       int32_t* grid_dimensions,
       double* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if (tidx < num_objects[0]) {
    result[3*tidx + 0] = timestep[0] * tractor_force[3*tidx + 0];
    result[3*tidx + 1] = timestep[0] * tractor_force[3*tidx + 1];
    result[3*tidx + 2] = timestep[0] * tractor_force[3*tidx + 2];
    // printf("cell_movement = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);
    
    double3 collision_force = make_double3(0, 0, 0);

    // Moore neighborhood
    int3 box_coords = get_box_coordinates_2(box_id[tidx], num_boxes_axis);
    for (int z = -1; z <= 1; z++) {
      for (int y = -1; y <= 1; y++) {
        for (int x = -1; x <= 1; x++) {
          uint32_t bidx = get_box_id_2(box_coords + make_int3(x, y, z), num_boxes_axis);
          if (lengths[bidx] != 0) {
            default_force(positions, diameters, tidx, starts[bidx], lengths[bidx], successors, squared_radius, &collision_force);
          }
        }
      }
    }

    // Mass needs to non-zero!
    double mh = timestep[0] / mass[tidx];
    // printf("mh = %f\n", mh);

    if (norm(collision_force) > adherence[tidx]) {
      result[3*tidx + 0] += collision_force.x * mh;
      result[3*tidx + 1] += collision_force.y * mh;
      result[3*tidx + 2] += collision_force.z * mh;
      // printf("collision_force = (%f, %f, %f)\n", collision_force.x, collision_force.y, collision_force.z);
      // printf("cell_movement (1) = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);

      if (norm(collision_force) * mh > max_displacement[0]) {
        result[3*tidx + 0] = max_displacement[0];
        result[3*tidx + 1] = max_displacement[0];
        result[3*tidx + 2] = max_displacement[0];
      }
      // printf("cell_movement (2) = (%f, %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx + 2]);
    }
  }
}

bdm::DisplacementOpCudaKernel::DisplacementOpCudaKernel(uint32_t num_objects, uint32_t num_boxes) {
  // printf("num_objects = %u  |  num_boxes = %u\n", num_objects, num_boxes);
  GpuErrchk(hipMalloc(&d_positions_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_diameters_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_tractor_force_, 3 * num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_adherence_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_box_id_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_mass_, num_objects * sizeof(double)));
  GpuErrchk(hipMalloc(&d_timestep_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_max_displacement_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_squared_radius_, sizeof(double)));
  GpuErrchk(hipMalloc(&d_num_objects_, sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t)));
  GpuErrchk(hipMalloc(&d_successors_, num_objects * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_box_length_, sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_num_boxes_axis_, 3 * sizeof(uint32_t)));
  GpuErrchk(hipMalloc(&d_grid_dimensions_, 3 * sizeof(int32_t)));
  GpuErrchk(hipMalloc(&d_cell_movements_, 3 * num_objects * sizeof(double)));
}

void bdm::DisplacementOpCudaKernel::LaunchDisplacementKernel(const double* positions,
    const double* diameters, const double* tractor_force, const double* adherence,
    uint32_t* box_id, const double* mass, const double* timestep, const double* max_displacement,
    const double* squared_radius, uint32_t* num_objects, uint32_t* starts,
    uint16_t* lengths, uint32_t* successors, uint32_t* box_length,
    uint32_t* num_boxes_axis, int32_t* grid_dimensions,
    double* cell_movements) {
  uint32_t num_boxes = num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

  GpuErrchk(hipMemcpy(d_positions_, 		positions, 3 * num_objects[0] * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_diameters_, 		diameters, num_objects[0] * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_tractor_force_, 	tractor_force, 3 * num_objects[0] * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_adherence_,     adherence, num_objects[0] * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_box_id_, 		box_id, num_objects[0] * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_mass_, 				mass, num_objects[0] * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_timestep_, 			timestep, sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_max_displacement_,  max_displacement, sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_squared_radius_, 	squared_radius, sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_num_objects_, 				num_objects, sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_starts_, 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_lengths_, 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_successors_, 		successors, num_objects[0] * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_box_length_, 		box_length, sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_num_boxes_axis_, 	num_boxes_axis, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpy(d_grid_dimensions_, 	grid_dimensions, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0, num_objects[0]);
  gridSize = (num_objects[0] + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  collide<<<gridSize, blockSize>>>(d_positions_, d_diameters_, d_tractor_force_,
    d_adherence_, d_box_id_, d_mass_, d_timestep_, d_max_displacement_,
    d_squared_radius_, d_num_objects_, d_starts_, d_lengths_, d_successors_,
    d_box_length_, d_num_boxes_axis_, d_grid_dimensions_, d_cell_movements_);

  // We need to wait for the kernel to finish before reading back the result
  hipDeviceSynchronize();
  hipMemcpy(cell_movements, d_cell_movements_, 3 * num_objects[0] * sizeof(double), hipMemcpyDeviceToHost);
}

void bdm::DisplacementOpCudaKernel::ResizeCellBuffers(uint32_t num_cells) {
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_successors_);
  hipFree(d_cell_movements_);

  hipMalloc(&d_positions_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_diameters_, num_cells * sizeof(double));
  hipMalloc(&d_tractor_force_, 3 * num_cells * sizeof(double));
  hipMalloc(&d_adherence_, num_cells * sizeof(double));
  hipMalloc(&d_box_id_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_mass_, num_cells * sizeof(double));
  hipMalloc(&d_successors_, num_cells * sizeof(uint32_t));
  hipMalloc(&d_cell_movements_, 3 * num_cells * sizeof(double));
}

void bdm::DisplacementOpCudaKernel::ResizeGridBuffers(uint32_t num_boxes) {
  hipFree(d_starts_);
  hipFree(d_lengths_);

  hipMalloc(&d_starts_, num_boxes * sizeof(uint32_t));
  hipMalloc(&d_lengths_, num_boxes * sizeof(uint16_t));
}

bdm::DisplacementOpCudaKernel::~DisplacementOpCudaKernel() {
  hipFree(d_positions_);
  hipFree(d_diameters_);
  hipFree(d_tractor_force_);
  hipFree(d_adherence_);
  hipFree(d_box_id_);
  hipFree(d_mass_);
  hipFree(d_timestep_);
  hipFree(d_max_displacement_);
  hipFree(d_squared_radius_);
  hipFree(d_num_objects_);
  hipFree(d_starts_);
  hipFree(d_lengths_);
  hipFree(d_successors_);
  hipFree(d_num_boxes_axis_);
  hipFree(d_grid_dimensions_);
  hipFree(d_cell_movements_);
}
