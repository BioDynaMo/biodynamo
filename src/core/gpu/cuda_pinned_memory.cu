// -----------------------------------------------------------------------------
//
// Copyright (C) The BioDynaMo Project.
// All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include "core/gpu/cuda_pinned_memory.h"

namespace bdm {

template <typename T>
void AllocPinned(T** d, uint64_t elements) {
  hipHostMalloc((void**)d, elements * sizeof(T));
}

template void AllocPinned<double>(double**, uint64_t);
template void AllocPinned<float>(float**, uint64_t);
template void AllocPinned<uint64_t>(uint64_t**, uint64_t);
template void AllocPinned<int64_t>(int64_t**, uint64_t);
template void AllocPinned<uint32_t>(uint32_t**, uint64_t);
template void AllocPinned<int32_t>(int32_t**, uint64_t);
template void AllocPinned<uint16_t>(uint16_t**, uint64_t);
template void AllocPinned<int16_t>(int16_t**, uint64_t);

}  // namespace bdm

