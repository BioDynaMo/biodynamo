// -----------------------------------------------------------------------------
//
// Copyright (C) 2021 CERN & University of Surrey for the benefit of the
// BioDynaMo collaboration. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include "core/gpu/cuda_pinned_memory.h"
#include "core/gpu/cuda_error_chk.h"

namespace bdm {

template <typename T>
void CudaAllocPinned(T** d, uint64_t elements) {
  GpuErrchk(hipHostMalloc((void**)d, elements * sizeof(T)));
}

template void CudaAllocPinned<double>(double**, uint64_t);
template void CudaAllocPinned<float>(float**, uint64_t);
template void CudaAllocPinned<uint64_t>(uint64_t**, uint64_t);
template void CudaAllocPinned<int64_t>(int64_t**, uint64_t);
template void CudaAllocPinned<uint32_t>(uint32_t**, uint64_t);
template void CudaAllocPinned<int32_t>(int32_t**, uint64_t);
template void CudaAllocPinned<uint16_t>(uint16_t**, uint64_t);
template void CudaAllocPinned<int16_t>(int16_t**, uint64_t);

void CudaFreePinned(void* p) {
  GpuErrchk(hipHostFree(p));
}

}  // namespace bdm
