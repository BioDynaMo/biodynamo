#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//
// Copyright (C) 2021 CERN & University of Surrey for the benefit of the
// BioDynaMo collaboration. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include "samples/common/inc/hip/hip_vector_types.h"

#include <unistd.h>
#include <iostream>

#include "core/gpu/cuda_error_chk.h"
#include "core/gpu/cuda_timer.h"
#include "core/gpu/helper_math_real_t.h"
#include "core/gpu/mechanical_forces_op_cuda_kernel.h"

void printMemoryUsage() {
  size_t availableMemory, totalMemory, usedMemory;
  hipMemGetInfo(&availableMemory, &totalMemory);
  usedMemory = totalMemory - availableMemory;
  std::cout << "Device memory: used " << usedMemory << " available "
            << availableMemory << " total " << totalMemory << std::endl;
}

__device__ real_t squared_euclidian_distance(real_t3* positions, uint32_t idx,
                                             uint32_t nidx) {
  auto diff = positions[idx] - positions[nidx];
  return dot(diff, diff);
}

__device__ int3 get_box_coordinates(uint32_t box_idx, uint3 num_boxes_axis) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis.x * num_boxes_axis.y);
  uint32_t remainder = box_idx % (num_boxes_axis.x * num_boxes_axis.y);
  box_coord.y = remainder / num_boxes_axis.x;
  box_coord.x = remainder % num_boxes_axis.x;
  return box_coord;
}

__device__ uint32_t get_box_id(int3 bc, uint3 num_boxes_axis) {
  return bc.z * num_boxes_axis.x * num_boxes_axis.y + bc.y * num_boxes_axis.x +
         bc.x;
}

__device__ void compute_force(real_t3* positions, real_t* diameters,
                              uint32_t idx, uint32_t nidx, real_t3* result) {
  real_t r1 = 0.5 * diameters[idx];
  real_t r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a
  // desired density.
  real_t additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  real_t3 comp = positions[idx] - positions[nidx];
  real_t center_distance = length(comp);

  // the overlap distance (how much one penetrates in the other)
  real_t delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    *result += make_real_t3(42.0, 42.0, 42.0);
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  real_t r = (r1 * r2) / (r1 + r2);
  real_t gamma = 1;  // attraction coeff
  real_t k = 2;      // repulsion coeff
  real_t f = k * delta - gamma * sqrt(r * delta);

  real_t module = f / center_distance;
  *result += module * comp;
  // printf("%f, %f, %f\n", module * comp.x, module * comp.y, module * comp.z);
  // printf(
  //     "Force between cell (%u) [%f, %f, %f] & cell (%u) [%f, %f, %f] = %f,%f, "
  //     "%f\n",
  //     idx, positions[idx].x, positions[idx].y, positions[idx].z, nidx,
  //     positions[nidx].x, positions[nidx].y, positions[nidx].z, module * comp.x,
  //     module * comp.y, module * comp.z);
}

__device__ void force(real_t3* positions, real_t* diameters, uint32_t idx,
                      uint32_t start, uint16_t length, uint32_t* successors,
                      real_t squared_radius, real_t3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (squared_euclidian_distance(positions, idx, nidx) < squared_radius) {
        compute_force(positions, diameters, idx, nidx, result);
      }
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__global__ void collide(real_t3* positions, real_t* diameters,
                        real_t3* tractor_force, real_t* adherence,
                        uint32_t* box_id, real_t* mass, real_t timestep,
                        real_t max_displacement, real_t squared_radius,
                        uint32_t num_agents, uint32_t* starts,
                        uint16_t* lengths, uint64_t* timestamps,
                        uint64_t current_timestamp, uint32_t* successors,
                        uint3* num_boxes_axis, real_t3* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  // if (tidx == 0) {
  //   printf("Positions = ");
  //   for (uint32_t i = 0; i < num_agents; i++) {
  //     printf("%f, %f, %f, ", positions[i].x, positions[i].y, positions[i].z);
  //   }
  //   printf("\nSuccessors = ");
  //   for (uint32_t i = 0; i < num_agents; i++) {
  //     printf("%u, ", successors[i]);
  //   }
  //   printf("\nbox_id = ");
  //   uint32_t num_boxes = num_boxes_axis->x * num_boxes_axis->y * num_boxes_axis->z;
  //   for (uint32_t i = 0; i < num_boxes; i++) {
  //     printf("%u, ", box_id[i]);
  //   }
  //   printf("\nTimestamps = ");
  //   for (uint32_t i = 0; i < num_boxes; i++) {
  //     printf("%u, ", timestamps[i]);
  //   }
  //   printf("\n");
  // }
  if (tidx < num_agents) {
    result[tidx] += timestep * tractor_force[tidx];

    real_t3 collision_force = make_real_t3(0, 0, 0);
    real_t3 movement_at_next_step = make_real_t3(0, 0, 0);

    // Moore neighborhood
    int3 box_coords = get_box_coordinates(box_id[tidx], *num_boxes_axis);
    for (int z = -1; z <= 1; z++) {
      for (int y = -1; y <= 1; y++) {
        for (int x = -1; x <= 1; x++) {
          uint32_t bidx =
              get_box_id(box_coords + make_int3(x, y, z), *num_boxes_axis);
          if (timestamps[bidx] == current_timestamp && lengths[bidx] != 0) {
            force(positions, diameters, tidx, starts[bidx], lengths[bidx],
                  successors, squared_radius, &collision_force);
          }
        }
      }
    }

    // Mass needs to non-zero!
    real_t mh = timestep / mass[tidx];
    // printf("mh = %f\n", mh);

    if (length(collision_force) > adherence[tidx]) {
      movement_at_next_step += collision_force * mh;
      // printf("collision_force = (%f, %f, %f)\n", collision_force.x,
      // collision_force.y, collision_force.z); printf("cell_movement (1) = (%f,
      // %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx +
      // 2]);

      if (length(collision_force) * mh > max_displacement) {
        movement_at_next_step = normalize(movement_at_next_step);
        movement_at_next_step *= max_displacement;
      }
      result[tidx] = movement_at_next_step;
      // printf("cell_movement (2) = (%f, %f, %f)\n", result[3*tidx + 0],
      // result[3*tidx + 1], result[3*tidx + 2]);
    }
  }
}

bdm::MechanicalForcesOpCudaKernel::MechanicalForcesOpCudaKernel(
    uint32_t num_agents, uint32_t num_boxes) {
  d_positions_.resize(num_agents);
  d_diameters_.resize(num_agents);
  d_tractor_force_.resize(num_agents);
  d_adherence_.resize(num_agents);
  d_box_id_.resize(num_agents);
  d_mass_.resize(num_agents);
  d_cell_movements_.resize(num_agents);
  d_successors_.resize(num_agents);
  d_starts_.resize(num_boxes);
  d_lengths_.resize(num_boxes);
  d_timestamps_.resize(num_boxes);
  d_num_boxes_axis_ = thrust::device_malloc<uint3>(1);
}

void bdm::MechanicalForcesOpCudaKernel::LaunchMechanicalForcesKernel(
    const real_t* positions, const real_t* diameters,
    const real_t* tractor_force, const real_t* adherence,
    const uint32_t* box_id, const real_t* mass, const real_t timestep,
    const real_t max_displacement, const real_t squared_radius,
    const uint32_t num_agents, uint32_t* starts, uint16_t* lengths,
    uint64_t* timestamps, uint64_t current_timestamp, uint32_t* successors,
    uint32_t* num_boxes_axis, real_t* cell_movements) {
  uint32_t num_boxes =
      num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

  // clang-format off
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_positions_.data()), 		positions, 3 * num_agents * sizeof(real_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_diameters_.data()), 		diameters, num_agents * sizeof(real_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_tractor_force_.data()), 	tractor_force, 3 * num_agents * sizeof(real_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_adherence_.data()),     adherence, num_agents * sizeof(real_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_box_id_.data()), 		box_id, num_agents * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_mass_.data()), 				mass, num_agents * sizeof(real_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_starts_.data()), 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_lengths_.data()), 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_timestamps_.data()), 			timestamps, num_boxes * sizeof(uint64_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_successors_.data()), 		successors, num_agents * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_num_boxes_axis_), 	num_boxes_axis, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));
  // clang-format on

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0,
                                     num_agents);
  gridSize = (num_agents + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  collide<<<gridSize, blockSize>>>(
      thrust::raw_pointer_cast(d_positions_.data()),
      thrust::raw_pointer_cast(d_diameters_.data()),
      thrust::raw_pointer_cast(d_tractor_force_.data()),
      thrust::raw_pointer_cast(d_adherence_.data()),
      thrust::raw_pointer_cast(d_box_id_.data()),
      thrust::raw_pointer_cast(d_mass_.data()), timestep, max_displacement,
      squared_radius, num_agents, thrust::raw_pointer_cast(d_starts_.data()),
      thrust::raw_pointer_cast(d_lengths_.data()),
      thrust::raw_pointer_cast(d_timestamps_.data()), current_timestamp,
      thrust::raw_pointer_cast(d_successors_.data()),
      thrust::raw_pointer_cast(d_num_boxes_axis_),
      thrust::raw_pointer_cast(d_cell_movements_.data()));

  hipMemcpyAsync(cell_movements,
                  thrust::raw_pointer_cast(d_cell_movements_.data()),
                  3 * num_agents * sizeof(real_t), hipMemcpyDeviceToHost);
}

void bdm::MechanicalForcesOpCudaKernel::Sync() const {
  hipDeviceSynchronize();
}

void bdm::MechanicalForcesOpCudaKernel::ResizeCellBuffers(uint32_t num_cells) {
  d_positions_.resize(num_cells);
  d_diameters_.resize(num_cells);
  d_tractor_force_.resize(num_cells);
  d_adherence_.resize(num_cells);
  d_box_id_.resize(num_cells);
  d_mass_.resize(num_cells);
  d_successors_.resize(num_cells);
  d_cell_movements_.resize(num_cells);
}

void bdm::MechanicalForcesOpCudaKernel::ResizeGridBuffers(uint32_t num_boxes) {
  d_starts_.resize(num_boxes);
  d_lengths_.resize(num_boxes);
  d_timestamps_.resize(num_boxes);
}

bdm::MechanicalForcesOpCudaKernel::~MechanicalForcesOpCudaKernel() {
  thrust::device_free(d_num_boxes_axis_);
}
