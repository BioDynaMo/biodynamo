#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//
// Copyright (C) 2021 CERN & University of Surrey for the benefit of the
// BioDynaMo collaboration. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
//
// See the LICENSE file distributed with this work for details.
// See the NOTICE file distributed with this work for additional information
// regarding copyright ownership.
//
// -----------------------------------------------------------------------------

#include <thrust/device_free.h>
#include <thrust/device_malloc.h>
#include "samples/common/inc/hip/hip_vector_types.h"

#include <unistd.h>
#include <iostream>

#include "core/gpu/cuda_error_chk.h"
#include "core/gpu/cuda_timer.h"
#include "core/gpu/helper_math_double.h"
#include "core/gpu/mechanical_forces_op_cuda_kernel.h"

void printMemoryUsage() {
  size_t availableMemory, totalMemory, usedMemory;
  hipMemGetInfo(&availableMemory, &totalMemory);
  usedMemory = totalMemory - availableMemory;
  std::cout << "Device memory: used " << usedMemory << " available "
            << availableMemory << " total " << totalMemory << std::endl;
}

__device__ double squared_euclidian_distance(double3* positions, uint32_t idx,
                                             uint32_t nidx) {
  auto diff = positions[idx] - positions[nidx];
  return dot(diff, diff);
}

__device__ int3 get_box_coordinates(uint32_t box_idx, uint3 num_boxes_axis) {
  int3 box_coord;
  box_coord.z = box_idx / (num_boxes_axis.x * num_boxes_axis.y);
  uint32_t remainder = box_idx % (num_boxes_axis.x * num_boxes_axis.y);
  box_coord.y = remainder / num_boxes_axis.x;
  box_coord.x = remainder % num_boxes_axis.x;
  return box_coord;
}

__device__ uint32_t get_box_id(int3 bc, uint3 num_boxes_axis) {
  return bc.z * num_boxes_axis.x * num_boxes_axis.y + bc.y * num_boxes_axis.x +
         bc.x;
}

__device__ void compute_force(double3* positions, double* diameters,
                              uint32_t idx, uint32_t nidx, double3* result) {
  double r1 = 0.5 * diameters[idx];
  double r2 = 0.5 * diameters[nidx];
  // We take virtual bigger radii to have a distant interaction, to get a
  // desired density.
  double additional_radius = 10.0 * 0.15;
  r1 += additional_radius;
  r2 += additional_radius;

  double3 comp = positions[idx] - positions[nidx];
  double center_distance = length(comp);

  // the overlap distance (how much one penetrates in the other)
  double delta = r1 + r2 - center_distance;

  if (delta < 0) {
    return;
  }

  // to avoid a division by 0 if the centers are (almost) at the same location
  if (center_distance < 0.00000001) {
    *result += make_double3(42.0, 42.0, 42.0);
    return;
  }

  // printf("Colliding cell [%d] and [%d]\n", idx, nidx);
  // printf("Delta for neighbor [%d] = %f\n", nidx, delta);

  // the force itself
  double r = (r1 * r2) / (r1 + r2);
  double gamma = 1;  // attraction coeff
  double k = 2;      // repulsion coeff
  double f = k * delta - gamma * sqrt(r * delta);

  double module = f / center_distance;
  *result += module * comp;
  // printf("%f, %f, %f\n", module * comp.x, module * comp.y, module * comp.z);
  // printf(
  //     "Force between cell (%u) [%f, %f, %f] & cell (%u) [%f, %f, %f] = %f,%f, "
  //     "%f\n",
  //     idx, positions[idx].x, positions[idx].y, positions[idx].z, nidx,
  //     positions[nidx].x, positions[nidx].y, positions[nidx].z, module * comp.x,
  //     module * comp.y, module * comp.z);
}

__device__ void force(double3* positions, double* diameters, uint32_t idx,
                      uint32_t start, uint16_t length, uint32_t* successors,
                      double squared_radius, double3* result) {
  uint32_t nidx = start;
  for (uint16_t nb = 0; nb < length; nb++) {
    if (nidx != idx) {
      if (squared_euclidian_distance(positions, idx, nidx) < squared_radius) {
        compute_force(positions, diameters, idx, nidx, result);
      }
    }
    // traverse linked-list
    nidx = successors[nidx];
  }
}

__global__ void collide(double3* positions, double* diameters,
                        double3* tractor_force, double* adherence,
                        uint32_t* box_id, double* mass, double timestep,
                        double max_displacement, double squared_radius,
                        uint32_t num_agents, uint32_t* starts,
                        uint16_t* lengths, uint64_t* timestamps,
                        uint64_t current_timestamp, uint32_t* successors,
                        uint3* num_boxes_axis, double3* result) {
  uint32_t tidx = blockIdx.x * blockDim.x + threadIdx.x;
  // if (tidx == 0) {
  //   printf("Positions = ");
  //   for (uint32_t i = 0; i < num_agents; i++) {
  //     printf("%f, %f, %f, ", positions[i].x, positions[i].y, positions[i].z);
  //   }
  //   printf("\nSuccessors = ");
  //   for (uint32_t i = 0; i < num_agents; i++) {
  //     printf("%u, ", successors[i]);
  //   }
  //   printf("\nbox_id = ");
  //   uint32_t num_boxes = num_boxes_axis->x * num_boxes_axis->y * num_boxes_axis->z;
  //   for (uint32_t i = 0; i < num_boxes; i++) {
  //     printf("%u, ", box_id[i]);
  //   }
  //   printf("\nTimestamps = ");
  //   for (uint32_t i = 0; i < num_boxes; i++) {
  //     printf("%u, ", timestamps[i]);
  //   }
  //   printf("\n");
  // }
  if (tidx < num_agents) {
    result[tidx] += timestep * tractor_force[tidx];

    double3 collision_force = make_double3(0, 0, 0);
    double3 movement_at_next_step = make_double3(0, 0, 0);

    // Moore neighborhood
    int3 box_coords = get_box_coordinates(box_id[tidx], *num_boxes_axis);
    for (int z = -1; z <= 1; z++) {
      for (int y = -1; y <= 1; y++) {
        for (int x = -1; x <= 1; x++) {
          uint32_t bidx =
              get_box_id(box_coords + make_int3(x, y, z), *num_boxes_axis);
          if (timestamps[bidx] == current_timestamp && lengths[bidx] != 0) {
            force(positions, diameters, tidx, starts[bidx], lengths[bidx],
                  successors, squared_radius, &collision_force);
          }
        }
      }
    }

    // Mass needs to non-zero!
    double mh = timestep / mass[tidx];
    // printf("mh = %f\n", mh);

    if (length(collision_force) > adherence[tidx]) {
      movement_at_next_step += collision_force * mh;
      // printf("collision_force = (%f, %f, %f)\n", collision_force.x,
      // collision_force.y, collision_force.z); printf("cell_movement (1) = (%f,
      // %f, %f)\n", result[3*tidx + 0], result[3*tidx + 1], result[3*tidx +
      // 2]);

      if (length(collision_force) * mh > max_displacement) {
        movement_at_next_step = normalize(movement_at_next_step);
        movement_at_next_step *= max_displacement;
      }
      result[tidx] = movement_at_next_step;
      // printf("cell_movement (2) = (%f, %f, %f)\n", result[3*tidx + 0],
      // result[3*tidx + 1], result[3*tidx + 2]);
    }
  }
}

bdm::MechanicalForcesOpCudaKernel::MechanicalForcesOpCudaKernel(
    uint32_t num_agents, uint32_t num_boxes) {
  d_positions_.resize(num_agents);
  d_diameters_.resize(num_agents);
  d_tractor_force_.resize(num_agents);
  d_adherence_.resize(num_agents);
  d_box_id_.resize(num_agents);
  d_mass_.resize(num_agents);
  d_cell_movements_.resize(num_agents);
  d_successors_.resize(num_agents);
  d_starts_.resize(num_boxes);
  d_lengths_.resize(num_boxes);
  d_timestamps_.resize(num_boxes);
  d_num_boxes_axis_ = thrust::device_malloc<uint3>(1);
}

void bdm::MechanicalForcesOpCudaKernel::LaunchMechanicalForcesKernel(
    const double* positions, const double* diameters,
    const double* tractor_force, const double* adherence,
    const uint32_t* box_id, const double* mass, const double timestep,
    const double max_displacement, const double squared_radius,
    const uint32_t num_agents, uint32_t* starts, uint16_t* lengths,
    uint64_t* timestamps, uint64_t current_timestamp, uint32_t* successors,
    uint32_t* num_boxes_axis, double* cell_movements) {
  uint32_t num_boxes =
      num_boxes_axis[0] * num_boxes_axis[1] * num_boxes_axis[2];

  // clang-format off
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_positions_.data()), 		positions, 3 * num_agents * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_diameters_.data()), 		diameters, num_agents * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_tractor_force_.data()), 	tractor_force, 3 * num_agents * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_adherence_.data()),     adherence, num_agents * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_box_id_.data()), 		box_id, num_agents * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_mass_.data()), 				mass, num_agents * sizeof(double), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_starts_.data()), 			starts, num_boxes * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_lengths_.data()), 			lengths, num_boxes * sizeof(uint16_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_timestamps_.data()), 			timestamps, num_boxes * sizeof(uint64_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_successors_.data()), 		successors, num_agents * sizeof(uint32_t), hipMemcpyHostToDevice));
  GpuErrchk(hipMemcpyAsync(thrust::raw_pointer_cast(d_num_boxes_axis_), 	num_boxes_axis, 3 * sizeof(uint32_t), hipMemcpyHostToDevice));
  // clang-format on

  int blockSize = 128;
  int minGridSize;
  int gridSize;

  // Get a near-optimal occupancy with the following thread organization
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, collide, 0,
                                     num_agents);
  gridSize = (num_agents + blockSize - 1) / blockSize;

  // printf("gridSize = %d  |  blockSize = %d\n", gridSize, blockSize);
  collide<<<gridSize, blockSize>>>(
      thrust::raw_pointer_cast(d_positions_.data()),
      thrust::raw_pointer_cast(d_diameters_.data()),
      thrust::raw_pointer_cast(d_tractor_force_.data()),
      thrust::raw_pointer_cast(d_adherence_.data()),
      thrust::raw_pointer_cast(d_box_id_.data()),
      thrust::raw_pointer_cast(d_mass_.data()), timestep, max_displacement,
      squared_radius, num_agents, thrust::raw_pointer_cast(d_starts_.data()),
      thrust::raw_pointer_cast(d_lengths_.data()),
      thrust::raw_pointer_cast(d_timestamps_.data()), current_timestamp,
      thrust::raw_pointer_cast(d_successors_.data()),
      thrust::raw_pointer_cast(d_num_boxes_axis_),
      thrust::raw_pointer_cast(d_cell_movements_.data()));

  hipMemcpyAsync(cell_movements,
                  thrust::raw_pointer_cast(d_cell_movements_.data()),
                  3 * num_agents * sizeof(double), hipMemcpyDeviceToHost);
}

void bdm::MechanicalForcesOpCudaKernel::Sync() const {
  hipDeviceSynchronize();
}

void bdm::MechanicalForcesOpCudaKernel::ResizeCellBuffers(uint32_t num_cells) {
  d_positions_.resize(num_cells);
  d_diameters_.resize(num_cells);
  d_tractor_force_.resize(num_cells);
  d_adherence_.resize(num_cells);
  d_box_id_.resize(num_cells);
  d_mass_.resize(num_cells);
  d_successors_.resize(num_cells);
  d_cell_movements_.resize(num_cells);
}

void bdm::MechanicalForcesOpCudaKernel::ResizeGridBuffers(uint32_t num_boxes) {
  d_starts_.resize(num_boxes);
  d_lengths_.resize(num_boxes);
  d_timestamps_.resize(num_boxes);
}

bdm::MechanicalForcesOpCudaKernel::~MechanicalForcesOpCudaKernel() {
  thrust::device_free(d_num_boxes_axis_);
}
